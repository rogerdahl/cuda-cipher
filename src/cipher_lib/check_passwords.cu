#include "hip/hip_runtime.h"
﻿// Stdlib.
#include "stdio.h"
#include <iostream>
#include <stdint.h>
#include <string.h>
#include <vector>

// CUDA
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

// App.
#include "check_passwords.h"
#include "cuda_utilities.h"
#include "int_types.h"

using namespace std;
using namespace thrust;

// Host.

__global__ void CheckPasswordsKernel(
    u32* found_password_index_out, u8* password_blocks, u32 n_passwords,
    u32 password_len, u8* pbkdf2_salt, u32 pbkdf2_salt_len, u32 pbkdf2_rounds,
    u32* blowfish_init_vector, u32* sha1_checksum, u8* contents_section,
    u32 contents_section_len);

int select_threads_per_block();

// PBKDF2-HMAC-SHA1

__device__ void PBKDF2(
    u8* key_out, const u8* password, u32 password_len, const u8* salt,
    u32 salt_len, u32 key_len, u32 rounds);

// HMAC-SHA1

__device__ void HMACSHA1(
    u32* hash_out, const void* key, u32 key_len, const void* msg, u32 msg_len);
__device__ void HMACSHA1InnerOuter(
    u32* hash_inner_out, u32* hash_outer_out, const void* key, u32 key_len);
__device__ void HMACSHA1Hash(
    u32* hash_out, u32* hash_inner, u32* hash_outer, const void* msg);
__device__ void HMACSHA1HashToBlock(u32* hash);

const u32 HMAC_INNER_PADDING(0x36363636);
const u32 HMAC_OUTER_PADDING(0x5c5c5c5c);

// SHA1

__device__ void SHA1InitState(u32* state);
__device__ void SHA1HashMessage(u32* hash, u8* msg, u32 len);
__device__ void SHA1CompressFullBlock(u32* state, const u32* block);
__device__ void SHA1CompressFullBlockHMACInner(u32* state, const u32* block);
__device__ void SHA1CompressFullBlockHMACOuter(u32* state, const u32* block);
__device__ void SHA1CompressLastBlock(u32* state, u8* msg, u32 len);
__device__ void SHA1CompressLastBlockEmpty(u32* state, u32 len);
__device__ void SHA1StateToHash(u32* hash, u32* state);
__device__ void SHA1CopyHash(u32* dst, u32* src);
__device__ bool SHA1CompareHash(u32* hash1, u32* hash2);
__device__ void SHA1SwapEndian(u32* hash);

const u32 N_SHA1_HASH_BYTES(20);
const u32 N_SHA1_HASH_WORDS(N_SHA1_HASH_BYTES / sizeof(u32));
const u32 N_SHA1_BLOCK_BYTES(64);
const u32 N_SHA1_BLOCK_WORDS(N_SHA1_BLOCK_BYTES / sizeof(u32));

// Blowfish

const u32 N_BLOWFISH_BLOCK_BYTES(8);
const u32 N_BLOWFISH_ROUNDS(16);

struct BlowfishSubkey
{
  u32 s0[256];
  u32 s1[256];
  u32 s2[256];
  u32 s3[256];
  u32 p[N_BLOWFISH_ROUNDS + 2];
};

__device__ void BlowfishCreateSubkey(
    BlowfishSubkey& subkey_out, u8* key, u32 key_len);
__device__ void BlowfishEncrypt(
    BlowfishSubkey& subkey, u32* left_out, u32* right_out);
//__device__ void BlowfishDecrypt(BlowfishSubkey& subkey, u32* left_out, u32*
//right_out); // not in use
__device__ void BlowfishDecryptBufferCFB(
    BlowfishSubkey& subkey, u32* buf_out, u32* buf_in, u32 n_blocks, u32* iv);

// Util, device

__device__ u32 SwapEndian32(u32 x);
__device__ void SwapEndian32Ptr(u32* x);
__device__ u32 Min(u32 a, u32 b);

// Util, host

void HostSwapEndian32Ptr(u32* x);
u32v u8vToLittleEndianu32v(u8v&);

// Given an array of passwords, check if any of them match the valid hash.
// All passwords in the array must have the same size.
void CheckPasswords(
    u32* found_password_index_out, u8v& password_blocks, u32 password_len,
    u8v& pbkdf2_salt, u32 pbkdf2_rounds, u8v& blowfish_init_vector,
    u8v& sha1_checksum, u8v& contents_section)
{
  // Copy passwords to device.
  host_vector<u8> password_blocks_h(
      password_blocks.begin(), password_blocks.end());
  device_vector<u8> password_blocks_d = password_blocks_h;
  u8* password_blocks_ptr_d(raw_pointer_cast(&password_blocks_d[0]));
  u32 n_passwords(password_blocks.size() / N_SHA1_BLOCK_BYTES);

  // Copy PBKDF2 salt to device (zero padded to N_SHA1_BLOCK_BYTES).
  host_vector<u8> pbkdf2_salt_h(pbkdf2_salt.begin(), pbkdf2_salt.end());
  pbkdf2_salt_h.resize(N_SHA1_BLOCK_BYTES);
  device_vector<u8> pbkdf2_salt_d = pbkdf2_salt_h;
  u8* pbkdf2_salt_ptr_d(raw_pointer_cast(&pbkdf2_salt_d[0]));

  // Convert Blowfish initialization vector to 2 32-bit little endian words and
  // copy to device.
  host_vector<u32> blowfish_init_vector_h(
      u8vToLittleEndianu32v(blowfish_init_vector));
  device_vector<u32> blowfish_init_vector_d = blowfish_init_vector_h;
  u32* blowfish_init_vector_ptr_d(raw_pointer_cast(&blowfish_init_vector_d[0]));

  // Convert SHA1 checksum to 5 32-bit little endian words and copy to device.
  host_vector<u32> sha1_checksum_h(u8vToLittleEndianu32v(sha1_checksum));
  // host_vector<u32> sha1_checksum_h(sha1_checksum);
  device_vector<u32> sha1_checksum_d = sha1_checksum_h;
  u32* sha1_checksum_ptr_d(raw_pointer_cast(&sha1_checksum_d[0]));

  // Copy first 1024 bytes of encrypted contents.xml to device.
  host_vector<u8> contents_section_h(
      contents_section.begin(), contents_section.end());
  device_vector<u8> contents_section_d = contents_section_h;
  u8* contents_section_ptr_d(raw_pointer_cast(&contents_section_d[0]));

  // Buffer for index of found password. Single 32 bit word.
  // device_vector<u32> found_password_index_out_d(1, 0xffffffff);
  // u32*
  // found_password_index_out_ptr_d(raw_pointer_cast(&found_password_index_out_d[0]));
  // device_vector bug workaround (crash when setting to fixed size or doing
  // resize)
  u32* found_password_index_out_ptr_d;
  u32 not_found(0xffffffff);
  cudaCheck(hipMalloc(&found_password_index_out_ptr_d, 4));
  cudaCheck(
      hipMemcpy(
          found_password_index_out_ptr_d, &not_found, sizeof(not_found),
          hipMemcpyHostToDevice));

  // Run kernel.

  // threads_per_block should be obtained from the occupancy calculator.

  //  for (u32 threads_per_block_x = 32; threads_per_block_x <= 1024;
  //  threads_per_block_x += 32) {
  u32 threads_per_block_x = select_threads_per_block();
  u32 threads_per_block_y(1);

  // Dimension of each thread block (number of threads to launch in each block).
  dim3 block_dim(threads_per_block_x, threads_per_block_y);

  // Dimension of the grid (number of blocks to launch).
  dim3 grid_dim(DivUp(n_passwords, threads_per_block_x));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(CheckPasswordsKernel), hipFuncCachePreferL1);

  hipDeviceSynchronize();
  hipEventRecord(start, 0);

  CheckPasswordsKernel<<<grid_dim, block_dim>>>(
      found_password_index_out_ptr_d, password_blocks_ptr_d, n_passwords,
      password_len, pbkdf2_salt_ptr_d, pbkdf2_salt.size(), pbkdf2_rounds,
      blowfish_init_vector_ptr_d, sha1_checksum_ptr_d, contents_section_ptr_d,
      contents_section.size());

  cudaCheckLastError("CheckPasswordsKernel");

  hipDeviceSynchronize();
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("time (ms): %f\n", elapsedTime);
  //}
  // host_vector<u32> found_password_index_out_h = found_password_index_out_d;
  //*found_password_index_out = found_password_index_out_h[0];
  // device_vector bug workaround (crash when setting to fixed size or doing
  // resize)
  cudaCheck(
      hipMemcpy(
          found_password_index_out, found_password_index_out_ptr_d, 4,
          hipMemcpyDeviceToHost));
}

int select_threads_per_block()
{
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  u32 threads_per_block = 0;

  // GeForce GTX 570
  if (prop.major == 2 && prop.minor == 0 && prop.multiProcessorCount == 15) {
    threads_per_block = 256;
  }
  // GeForce GTX 660 (3.0)
  else if (prop.major == 3 && prop.minor == 0 && prop.multiProcessorCount == 5) {
    threads_per_block = 256;
  }
  // GeForce GTX 780 Ti (3.5)
  else if (prop.major == 3 && prop.minor == 5 && prop.multiProcessorCount == 15) {
    threads_per_block = 768;
  }
  // GeForce GTX 750 Ti (5.0)
  else if (prop.major == 5 && prop.minor == 0 && prop.multiProcessorCount == 5) {
    threads_per_block = 512;
  }
  else {
    threads_per_block = 512;
  }
  printf("threads per block: %d\n", threads_per_block);

  return threads_per_block;
}

// passwords: array of passwords. Each is zero padded out to 64 bytes.
// key: 64 bytes, zero padded
// target_sha1_hash: 5 u32 words.
// found_password_idx: 1 u32 word (output)
__global__ void CheckPasswordsKernel(
    u32* found_password_index_out, u8* password_blocks, u32 n_passwords,
    u32 password_len, u8* pbkdf2_salt, u32 pbkdf2_salt_len, u32 pbkdf2_rounds,
    u32* blowfish_init_vector, u32* sha1_checksum, u8* contents_section,
    u32 contents_section_len)
{
  u32 i(blockIdx.x * blockDim.x + threadIdx.x);
  if (i >= n_passwords) {
    return;
  }

  // Get SHA1 hash of the password.
  u32 hash[N_SHA1_HASH_WORDS];
  SHA1HashMessage(hash, password_blocks + i * N_SHA1_BLOCK_BYTES, password_len);
  SHA1SwapEndian(hash);

  // Use PBKDF2 key extender to derive Blowfish key from password and salt.
  u8 key[16];
  PBKDF2(
      key, (u8*)hash, N_SHA1_HASH_BYTES, pbkdf2_salt, pbkdf2_salt_len,
      sizeof(key), pbkdf2_rounds);

  //// For benchmarking only the PBKDF2-HMAC-SHA1, make sure the compiler can't
  ///drop the PBKDF2 calculation.
  //// Only stores k in the extremely unlikely case that the 128-bit key is all
  ///zeroes.
  // u8 k = 0;
  // for (int j = 0; j < 16; ++j) {
  //  k |= key[j];
  //}
  // if (!k) {
  //  *found_password_index_out = i;
  //}
  // return;

  // Blowfish decrypt first 1024 bytes of contents.xml with blowfish key and
  // initialization vector.
  BlowfishSubkey subkey;
  BlowfishCreateSubkey(subkey, key, sizeof(key));
  u32 n_blowfish_blocks(contents_section_len / N_BLOWFISH_BLOCK_BYTES);
  u32 contents_section_decrypted[1024 / sizeof(u32)];
  BlowfishDecryptBufferCFB(
      subkey, contents_section_decrypted, (u32*)contents_section,
      n_blowfish_blocks, (u32*)blowfish_init_vector);

  // Get SHA1 hash of decrypted version of first 1024 bytes of contents.xml.
  SHA1HashMessage(hash, (u8*)contents_section_decrypted, contents_section_len);

  // If SHA1 hash matches the one supplied in the OpenOffice document manifest,
  // the password
  // has been found.
  if (SHA1CompareHash(hash, sha1_checksum)) {
    *found_password_index_out = i;
  }
}

///////////////////////////////////////////////////////////////////////////
// PBKDF2

// Inner loop: YES
__device__ void PBKDF2(
    u8* key_out, const u8* password, u32 password_len, const u8* salt,
    u32 salt_len, u32 key_len, u32 rounds)
{
  u32 inner[N_SHA1_HASH_WORDS];
  u32 outer[N_SHA1_HASH_WORDS];
  HMACSHA1InnerOuter(inner, outer, password, password_len);

  u8 salt_local
      [32]; /////////////////////////////////////////////////////////////////////////////
            ///arbitrary
  memcpy(salt_local, salt, salt_len);

  for (int count(1); key_len > 0; ++count) {
    salt_local[salt_len + 0] = (count >> 24) & 0xff;
    salt_local[salt_len + 1] = (count >> 16) & 0xff;
    salt_local[salt_len + 2] = (count >> 8) & 0xff;
    salt_local[salt_len + 3] = count & 0xff;

    u32 state[64 / sizeof(u32)];
    HMACSHA1(state, password, password_len, salt_local, salt_len + 4);
    u32 obuf[N_SHA1_HASH_WORDS];
    SHA1CopyHash(obuf, state);
    HMACSHA1HashToBlock(state);

    // INNER LOOP.
    for (int i(1); i < rounds; ++i) {
      HMACSHA1Hash(state, inner, outer, state);
      for (int j(0); j < N_SHA1_HASH_WORDS; ++j) {
        obuf[j] ^= state[j];
      }
    }

    u32 r(Min(key_len, N_SHA1_HASH_BYTES));
    memcpy(key_out, obuf, r);
    key_out += r;
    key_len -= r;
  };
}

///////////////////////////////////////////////////////////////////////////
// HMAC-SHA1

// Creates a SHA1 hash from a message and a key. The difference from
// regular SHA1 is the presense of the key.
//
// hash_out: The generated SHA1 hash. 20 bytes.
// key must be a 5 * 4 (20) byte hash.
// key_len must be <= N_SHA1_BLOCK_BYTES.
// key = password
// msg = salt (first round), sha1_hash (remaining rounds)
// len of openoffice password = 1 or more bytes
// len of openoffice salt = 16 bytes
// len of sha1_hash = 20 bytes
//
// Inner loop: NO
__device__ void HMACSHA1(
    u32* hash_out, const void* key, u32 key_len, const void* msg, u32 msg_len)
{
  u32 state[N_SHA1_HASH_WORDS];

  u8 __align__(4) buffer[N_SHA1_BLOCK_BYTES];
  memset(buffer, 0, N_SHA1_BLOCK_BYTES);
  memcpy(buffer, key, key_len);

  SHA1InitState(state);
  SHA1CompressFullBlockHMACInner(state, (u32*)buffer);

  while (msg_len >= N_SHA1_BLOCK_BYTES) {
    SHA1CompressFullBlock(state, (u32*)msg);
    msg = (u8*)msg + N_SHA1_BLOCK_BYTES;
    msg_len -= N_SHA1_BLOCK_BYTES;
  }

  // N_SHA1_BLOCK_BYTES is added to msg_len because message is being
  // (virtually) appended to i_key_pad.
  SHA1CompressLastBlock(state, (u8*)msg, N_SHA1_BLOCK_BYTES + msg_len);

  u32 hash[N_SHA1_HASH_WORDS];
  SHA1StateToHash(hash, state);
  SHA1InitState(hash_out);
  SHA1CompressFullBlockHMACOuter(hash_out, (u32*)buffer);
  SHA1CompressLastBlock(hash_out, (u8*)hash, N_SHA1_BLOCK_BYTES + 5 * 4);
  SHA1SwapEndian(hash_out);
}

// Optimization: Allows computation of the some hashes to be moved outside of
// the inner loop in PBKDF2.
// Inner loop: NO
__device__ void HMACSHA1InnerOuter(
    u32* hash_inner_out, u32* hash_outer_out, const void* key, u32 key_len)
{
  u8 __align__(4) buffer[N_SHA1_BLOCK_BYTES];
  memset(buffer, 0, N_SHA1_BLOCK_BYTES);
  memcpy(buffer, key, key_len);

  SHA1InitState(hash_inner_out);
  SHA1CompressFullBlockHMACInner(hash_inner_out, (u32*)buffer);

  SHA1InitState(hash_outer_out);
  SHA1CompressFullBlockHMACOuter(hash_outer_out, (u32*)buffer);
}

// Optimized for msg being an SHA1 hash (20 bytes).
// Inner loop: YES
__device__ void HMACSHA1Hash(
    u32* hash_out, u32* hash_inner, u32* hash_outer, const void* msg)
{
  u32 state[N_SHA1_HASH_WORDS];
  SHA1CopyHash(state, hash_inner);
  SHA1CompressFullBlock(state, reinterpret_cast<const u32*>(msg));
  u32 hash[64 / sizeof(u32)];
  SHA1StateToHash(hash, state);
  SHA1CopyHash(hash_out, hash_outer);
  HMACSHA1HashToBlock(hash);
  SHA1CompressFullBlock(hash_out, reinterpret_cast<u32*>(hash));
  SHA1SwapEndian(hash_out);
}

// Prepare a 64 byte block that contains a 20 byte SHA1 hash to be hashed as
// the last block in a two-block SHA1 message.
// Inner loop: YES
__device__ void HMACSHA1HashToBlock(u32* hash)
{
  // Set message terminating bit.
  hash[5] = 0x00000080; // SwapEndian32(0x80000000);
  hash[6] = 0x00000000;
  hash[7] = 0x00000000;
  hash[8] = 0x00000000;
  hash[9] = 0x00000000;
  hash[10] = 0x00000000;
  hash[11] = 0x00000000;
  hash[12] = 0x00000000;
  hash[13] = 0x00000000;
  // Set message size (1st block 64 bytes, second block 20 bytes = 84 bytes =
  // 672 bits = 02a0)
  hash[14] = 0x00000000;
  hash[15] = 0xa0020000; // SwapEndian32(0x000002a0);
}

///////////////////////////////////////////////////////////////////////////////
// SHA1

// Inner loop: YES
__device__ void SHA1InitState(u32* state)
{
  state[0] = 0x67452301;
  state[1] = 0xefcdab89;
  state[2] = 0x98badcfe;
  state[3] = 0x10325476;
  state[4] = 0xc3d2e1f0;
}

// Inner loop: NO
__device__ void SHA1HashMessage(u32* hash, u8* msg, u32 len)
{
  SHA1InitState(hash);

  for (int i(0); i + N_SHA1_BLOCK_BYTES <= len; i += N_SHA1_BLOCK_BYTES) {
    SHA1CompressFullBlock(hash, reinterpret_cast<u32*>(msg + i));
  }

  SHA1CompressLastBlock(hash, msg, len);

  // SHA1SwapEndian(hash);
}

// An SHA1 block is 16 32-bit words = 64 bytes = 512 bits.
// An SHA1 hash is 5 32-bit words = 20 bytes = 160 bits.
#define SCHEDULE(i)                                            \
  tmp = schedule[(i - 3) & 0xf] ^ schedule[(i - 8) & 0xf]      \
        ^ schedule[(i - 14) & 0xf] ^ schedule[(i - 16) & 0xf]; \
  schedule[i & 0xf] = tmp << 1 | tmp >> 31;
#define R0A(a, b, c, d, e, i)                                    \
  schedule[i] = (block[i] << 24) | ((block[i] & 0xff00) << 8)    \
                | ((block[i] >> 8) & 0xff00) | (block[i] >> 24); \
  RTAIL(a, b, e, ((b & c) | (~b & d)), i, 0x5a827999)
#define R0B(a, b, c, d, e, i) \
  SCHEDULE(i) RTAIL(a, b, e, ((b & c) | (~b & d)), i, 0x5a827999)
#define R1(a, b, c, d, e, i) \
  SCHEDULE(i) RTAIL(a, b, e, (b ^ c ^ d), i, 0x6ed9eba1)
#define R2(a, b, c, d, e, i) \
  SCHEDULE(i) RTAIL(a, b, e, ((b & c) ^ (b & d) ^ (c & d)), i, 0x8f1bbcdc)
#define R3(a, b, c, d, e, i) \
  SCHEDULE(i) RTAIL(a, b, e, (b ^ c ^ d), i, 0xca62c1d6)
#define RTAIL(a, b, e, f, i, k)                        \
  e += (a << 5 | a >> 27) + f + k + schedule[i & 0xf]; \
  b = b << 30 | b >> 2;
#define RS()                                                                                \
  R0B(e, a, b, c, d, 16)                                                                    \
  R0B(d, e, a, b, c, 17) R0B(c, d, e, a, b, 18) R0B(b, c, d, e, a, 19) R1(                  \
      a, b, c, d, e, 20) R1(e, a, b, c, d, 21) R1(d, e, a, b, c, 22)                        \
      R1(c, d, e, a, b, 23) R1(b, c, d, e, a, 24) R1(a, b, c, d, e, 25) R1(                 \
          e, a, b, c, d, 26) R1(d, e, a, b, c, 27) R1(c, d, e, a, b, 28)                    \
          R1(b, c, d, e, a, 29) R1(a, b, c, d, e, 30) R1(                                   \
              e,                                                                            \
              a,                                                                            \
              b,                                                                            \
              c,                                                                            \
              d,                                                                            \
              31) R1(d, e, a, b, c, 32)                                                     \
              R1(c, d, e, a, b, 33) R1(                                                     \
                  b,                                                                        \
                  c,                                                                        \
                  d,                                                                        \
                  e,                                                                        \
                  a,                                                                        \
                  34) R1(a, b, c, d, e, 35)                                                 \
                  R1(e, a, b, c, d, 36) R1(                                                 \
                      d,                                                                    \
                      e,                                                                    \
                      a,                                                                    \
                      b,                                                                    \
                      c,                                                                    \
                      37) R1(c, d, e, a, b, 38)                                             \
                      R1(b, c, d, e, a, 39) R2(                                             \
                          a,                                                                \
                          b,                                                                \
                          c,                                                                \
                          d,                                                                \
                          e,                                                                \
                          40) R2(e, a, b, c, d, 41)                                         \
                          R2(d, e, a, b, c, 42) R2(                                         \
                              c,                                                            \
                              d,                                                            \
                              e,                                                            \
                              a,                                                            \
                              b,                                                            \
                              43) R2(b, c, d, e, a, 44)                                     \
                              R2(a, b, c, d, e, 45) R2(                                     \
                                  e,                                                        \
                                  a,                                                        \
                                  b,                                                        \
                                  c,                                                        \
                                  d,                                                        \
                                  46) R2(d, e, a, b, c, 47)                                 \
                                  R2(c, d, e, a, b, 48) R2(b, c, d, e, a, 49) R2(           \
                                      a,                                                    \
                                      b,                                                    \
                                      c,                                                    \
                                      d,                                                    \
                                      e,                                                    \
                                      50) R2(e, a, b, c, d, 51)                             \
                                      R2(d, e, a, b, c, 52) R2(c, d, e, a, b, 53) R2(       \
                                          b,                                                \
                                          c,                                                \
                                          d,                                                \
                                          e,                                                \
                                          a,                                                \
                                          54) R2(a, b, c, d, e, 55)                         \
                                          R2(e, a, b, c, d, 56) R2(d, e, a, b, c, 57) R2(   \
                                              c,                                            \
                                              d,                                            \
                                              e,                                            \
                                              a,                                            \
                                              b,                                            \
                                              58) R2(b, c, d, e, a, 59)                     \
                                              R3(a, b, c, d, e, 60) R3(                     \
                                                  e,                                        \
                                                  a,                                        \
                                                  b,                                        \
                                                  c,                                        \
                                                  d,                                        \
                                                  61) R3(d, e, a, b, c, 62)                 \
                                                  R3(c, d, e, a, b, 63) R3(                 \
                                                      b,                                    \
                                                      c,                                    \
                                                      d,                                    \
                                                      e,                                    \
                                                      a,                                    \
                                                      64) R3(a, b, c, d, e, 65)             \
                                                      R3(e, a, b, c, d, 66) R3(             \
                                                          d,                                \
                                                          e,                                \
                                                          a,                                \
                                                          b,                                \
                                                          c,                                \
                                                          67) R3(c, d, e, a, b, 68)         \
                                                          R3(b, c, d, e, a, 69) R3(         \
                                                              a,                            \
                                                              b,                            \
                                                              c,                            \
                                                              d,                            \
                                                              e,                            \
                                                              70) R3(e, a, b, c, d, 71)     \
                                                              R3(d, e, a, b, c, 72) R3(     \
                                                                  c,                        \
                                                                  d,                        \
                                                                  e,                        \
                                                                  a,                        \
                                                                  b,                        \
                                                                  73) R3(b, c, d, e, a, 74) \
                                                                  R3(a, b, c, d, e, 75) R3( \
                                                                      e,                    \
                                                                      a,                    \
                                                                      b,                    \
                                                                      c,                    \
                                                                      d,                    \
                                                                      76)                   \
                                                                      R3(d, e,              \
                                                                         a, b,              \
                                                                         c,                 \
                                                                         77)                \
                                                                          R3(c,             \
                                                                             d,             \
                                                                             e,             \
                                                                             a,             \
                                                                             b,             \
                                                                             78)            \
                                                                              R3(b,         \
                                                                                 c,         \
                                                                                 d,         \
                                                                                 e,         \
                                                                                 a,         \
                                                                                 79)

// Inner loop: YES
__device__ void SHA1CompressFullBlock(u32* state, const u32* block)
{
  u32 a(state[0]);
  u32 b(state[1]);
  u32 c(state[2]);
  u32 d(state[3]);
  u32 e(state[4]);
  u32 schedule[16];
  u32 tmp;
  R0A(a, b, c, d, e, 0)
  R0A(e, a, b, c, d, 1) R0A(d, e, a, b, c, 2) R0A(c, d, e, a, b, 3)
      R0A(b, c, d, e, a, 4) R0A(a, b, c, d, e, 5) R0A(e, a, b, c, d, 6)
          R0A(d, e, a, b, c, 7) R0A(c, d, e, a, b, 8) R0A(b, c, d, e, a, 9)
              R0A(a, b, c, d, e, 10) R0A(e, a, b, c, d, 11)
                  R0A(d, e, a, b, c, 12) R0A(c, d, e, a, b, 13)
                      R0A(b, c, d, e, a, 14) R0A(a, b, c, d, e, 15) RS()
                          state[0] += a;
  state[1] += b;
  state[2] += c;
  state[3] += d;
  state[4] += e;
}

#define R0AHMACI(a, b, c, d, e, i)                                          \
  tmp = block[i] ^ HMAC_INNER_PADDING;                                      \
  schedule[i] = (tmp << 24) | ((tmp & 0xff00) << 8) | ((tmp >> 8) & 0xff00) \
                | (tmp >> 24);                                              \
  RTAIL(a, b, e, ((b & c) | (~b & d)), i, 0x5a827999)

// Inner loop: NO
__device__ void SHA1CompressFullBlockHMACInner(u32* state, const u32* block)
{
  u32 a(state[0]);
  u32 b(state[1]);
  u32 c(state[2]);
  u32 d(state[3]);
  u32 e(state[4]);
  u32 schedule[16];
  u32 tmp;
  R0AHMACI(a, b, c, d, e, 0)
  R0AHMACI(e, a, b, c, d, 1) R0AHMACI(d, e, a, b, c, 2)
      R0AHMACI(c, d, e, a, b, 3) R0AHMACI(b, c, d, e, a, 4)
          R0AHMACI(a, b, c, d, e, 5) R0AHMACI(e, a, b, c, d, 6)
              R0AHMACI(d, e, a, b, c, 7) R0AHMACI(c, d, e, a, b, 8)
                  R0AHMACI(b, c, d, e, a, 9) R0AHMACI(a, b, c, d, e, 10)
                      R0AHMACI(e, a, b, c, d, 11) R0AHMACI(d, e, a, b, c, 12)
                          R0AHMACI(c, d, e, a, b, 13)
                              R0AHMACI(b, c, d, e, a, 14)
                                  R0AHMACI(a, b, c, d, e, 15) RS() state[0] +=
      a;
  state[1] += b;
  state[2] += c;
  state[3] += d;
  state[4] += e;
}

#define R0AHMACO(a, b, c, d, e, i)                                          \
  tmp = block[i] ^ HMAC_OUTER_PADDING;                                      \
  schedule[i] = (tmp << 24) | ((tmp & 0xff00) << 8) | ((tmp >> 8) & 0xff00) \
                | (tmp >> 24);                                              \
  RTAIL(a, b, e, ((b & c) | (~b & d)), i, 0x5a827999)

// Inner loop: NO
__device__ void SHA1CompressFullBlockHMACOuter(u32* state, const u32* block)
{
  u32 a(state[0]);
  u32 b(state[1]);
  u32 c(state[2]);
  u32 d(state[3]);
  u32 e(state[4]);
  u32 schedule[16];
  u32 tmp;
  R0AHMACO(a, b, c, d, e, 0)
  R0AHMACO(e, a, b, c, d, 1) R0AHMACO(d, e, a, b, c, 2)
      R0AHMACO(c, d, e, a, b, 3) R0AHMACO(b, c, d, e, a, 4)
          R0AHMACO(a, b, c, d, e, 5) R0AHMACO(e, a, b, c, d, 6)
              R0AHMACO(d, e, a, b, c, 7) R0AHMACO(c, d, e, a, b, 8)
                  R0AHMACO(b, c, d, e, a, 9) R0AHMACO(a, b, c, d, e, 10)
                      R0AHMACO(e, a, b, c, d, 11) R0AHMACO(d, e, a, b, c, 12)
                          R0AHMACO(c, d, e, a, b, 13)
                              R0AHMACO(b, c, d, e, a, 14)
                                  R0AHMACO(a, b, c, d, e, 15) RS() state[0] +=
      a;
  state[1] += b;
  state[2] += c;
  state[3] += d;
  state[4] += e;
}

// Process the last, partial block and message length.
// The SHA1 spec requires the message to end with a "1" bit and then 8 bytes
// with the length of the message, in bits.
// Inner loop: NO
__device__ void SHA1CompressLastBlock(u32* state, u8* msg, u32 len)
{
  u32 block[N_SHA1_BLOCK_WORDS];
  u8* byteBlock = (u8*)block;

  int rem = len % 64;
  memcpy(byteBlock, msg, rem);

  byteBlock[rem] = 0x80;
  rem++;
  if (N_SHA1_BLOCK_BYTES - rem >= 8) {
    memset(byteBlock + rem, 0, 56 - rem);
  }
  else {
    memset(byteBlock + rem, 0, N_SHA1_BLOCK_BYTES - rem);
    SHA1CompressFullBlock(state, block);
    memset(block, 0, 56);
  }

  u64 longLen = ((u64)len) << 3;
  for (int i = 0; i < 8; i++) {
    byteBlock[N_SHA1_BLOCK_BYTES - 1 - i] = (u8)(longLen >> (i * 8));
  }
  SHA1CompressFullBlock(state, block);
}

// Special case for hashing the last block, when the last block is empty.
// This becomes the last step in hashing any message of length that is a
// multiple of 64 bytes.
// Inner loop: NO
__device__ void SHA1CompressLastBlockEmpty(u32* state, u32 len)
{
  u32 block[N_SHA1_BLOCK_WORDS] = { 0 };
  u8* byteBlock = (u8*)block;
  byteBlock[0] = 0x80;

  u64 long_len(len * 8);
  for (int i(0); i < 8; i++) {
    byteBlock[N_SHA1_BLOCK_BYTES - 1 - i] = (u8)(long_len >> (i * 8));
  }
  SHA1CompressFullBlock(state, block);
}

// Inner loop: YES
__device__ void SHA1StateToHash(u32* hash, u32* state)
{
  for (int i(0); i < N_SHA1_HASH_WORDS; ++i) {
    hash[i] = SwapEndian32(state[i]);
  }
}

// Inner loop: YES
__device__ void SHA1CopyHash(u32* dst, u32* src)
{
  for (int i(0); i < N_SHA1_HASH_WORDS; ++i) {
    dst[i] = src[i];
  }
}

__device__ bool SHA1CompareHash(u32* hash1, u32* hash2)
{
  return !(
      hash1[0] != hash2[0] || hash1[1] != hash2[1] || hash1[2] != hash2[2]
      || hash1[3] != hash2[3] || hash1[4] != hash2[4]);
}

// Inner loop: YES
__device__ void SHA1SwapEndian(u32* hash)
{
  for (int i(0); i < N_SHA1_HASH_WORDS; ++i) {
    SwapEndian32Ptr(hash + i);
  }
}

///////////////////////////////////////////////////////////////////////////////
// Blowfish

// Substitution-boxes.
__constant__ const u32 ks0[256] =
    { 0xD1310BA6, 0x98DFB5AC, 0x2FFD72DB, 0xD01ADFB7, 0xB8E1AFED, 0x6A267E96,
      0xBA7C9045, 0xF12C7F99, 0x24A19947, 0xB3916CF7, 0x0801F2E2, 0x858EFC16,
      0x636920D8, 0x71574E69, 0xA458FEA3, 0xF4933D7E, 0x0D95748F, 0x728EB658,
      0x718BCD58, 0x82154AEE, 0x7B54A41D, 0xC25A59B5, 0x9C30D539, 0x2AF26013,
      0xC5D1B023, 0x286085F0, 0xCA417918, 0xB8DB38EF, 0x8E79DCB0, 0x603A180E,
      0x6C9E0E8B, 0xB01E8A3E, 0xD71577C1, 0xBD314B27, 0x78AF2FDA, 0x55605C60,
      0xE65525F3, 0xAA55AB94, 0x57489862, 0x63E81440, 0x55CA396A, 0x2AAB10B6,
      0xB4CC5C34, 0x1141E8CE, 0xA15486AF, 0x7C72E993, 0xB3EE1411, 0x636FBC2A,
      0x2BA9C55D, 0x741831F6, 0xCE5C3E16, 0x9B87931E, 0xAFD6BA33, 0x6C24CF5C,
      0x7A325381, 0x28958677, 0x3B8F4898, 0x6B4BB9AF, 0xC4BFE81B, 0x66282193,
      0x61D809CC, 0xFB21A991, 0x487CAC60, 0x5DEC8032, 0xEF845D5D, 0xE98575B1,
      0xDC262302, 0xEB651B88, 0x23893E81, 0xD396ACC5, 0x0F6D6FF3, 0x83F44239,
      0x2E0B4482, 0xA4842004, 0x69C8F04A, 0x9E1F9B5E, 0x21C66842, 0xF6E96C9A,
      0x670C9C61, 0xABD388F0, 0x6A51A0D2, 0xD8542F68, 0x960FA728, 0xAB5133A3,
      0x6EEF0B6C, 0x137A3BE4, 0xBA3BF050, 0x7EFB2A98, 0xA1F1651D, 0x39AF0176,
      0x66CA593E, 0x82430E88, 0x8CEE8619, 0x456F9FB4, 0x7D84A5C3, 0x3B8B5EBE,
      0xE06F75D8, 0x85C12073, 0x401A449F, 0x56C16AA6, 0x4ED3AA62, 0x363F7706,
      0x1BFEDF72, 0x429B023D, 0x37D0D724, 0xD00A1248, 0xDB0FEAD3, 0x49F1C09B,
      0x075372C9, 0x80991B7B, 0x25D479D8, 0xF6E8DEF7, 0xE3FE501A, 0xB6794C3B,
      0x976CE0BD, 0x04C006BA, 0xC1A94FB6, 0x409F60C4, 0x5E5C9EC2, 0x196A2463,
      0x68FB6FAF, 0x3E6C53B5, 0x1339B2EB, 0x3B52EC6F, 0x6DFC511F, 0x9B30952C,
      0xCC814544, 0xAF5EBD09, 0xBEE3D004, 0xDE334AFD, 0x660F2807, 0x192E4BB3,
      0xC0CBA857, 0x45C8740F, 0xD20B5F39, 0xB9D3FBDB, 0x5579C0BD, 0x1A60320A,
      0xD6A100C6, 0x402C7279, 0x679F25FE, 0xFB1FA3CC, 0x8EA5E9F8, 0xDB3222F8,
      0x3C7516DF, 0xFD616B15, 0x2F501EC8, 0xAD0552AB, 0x323DB5FA, 0xFD238760,
      0x53317B48, 0x3E00DF82, 0x9E5C57BB, 0xCA6F8CA0, 0x1A87562E, 0xDF1769DB,
      0xD542A8F6, 0x287EFFC3, 0xAC6732C6, 0x8C4F5573, 0x695B27B0, 0xBBCA58C8,
      0xE1FFA35D, 0xB8F011A0, 0x10FA3D98, 0xFD2183B8, 0x4AFCB56C, 0x2DD1D35B,
      0x9A53E479, 0xB6F84565, 0xD28E49BC, 0x4BFB9790, 0xE1DDF2DA, 0xA4CB7E33,
      0x62FB1341, 0xCEE4C6E8, 0xEF20CADA, 0x36774C01, 0xD07E9EFE, 0x2BF11FB4,
      0x95DBDA4D, 0xAE909198, 0xEAAD8E71, 0x6B93D5A0, 0xD08ED1D0, 0xAFC725E0,
      0x8E3C5B2F, 0x8E7594B7, 0x8FF6E2FB, 0xF2122B64, 0x8888B812, 0x900DF01C,
      0x4FAD5EA0, 0x688FC31C, 0xD1CFF191, 0xB3A8C1AD, 0x2F2F2218, 0xBE0E1777,
      0xEA752DFE, 0x8B021FA1, 0xE5A0CC0F, 0xB56F74E8, 0x18ACF3D6, 0xCE89E299,
      0xB4A84FE0, 0xFD13E0B7, 0x7CC43B81, 0xD2ADA8D9, 0x165FA266, 0x80957705,
      0x93CC7314, 0x211A1477, 0xE6AD2065, 0x77B5FA86, 0xC75442F5, 0xFB9D35CF,
      0xEBCDAF0C, 0x7B3E89A0, 0xD6411BD3, 0xAE1E7E49, 0x00250E2D, 0x2071B35E,
      0x226800BB, 0x57B8E0AF, 0x2464369B, 0xF009B91E, 0x5563911D, 0x59DFA6AA,
      0x78C14389, 0xD95A537F, 0x207D5BA2, 0x02E5B9C5, 0x83260376, 0x6295CFA9,
      0x11C81968, 0x4E734A41, 0xB3472DCA, 0x7B14A94A, 0x1B510052, 0x9A532915,
      0xD60F573F, 0xBC9BC6E4, 0x2B60A476, 0x81E67400, 0x08BA6FB5, 0x571BE91F,
      0xF296EC6B, 0x2A0DD915, 0xB6636521, 0xE7B9F9B6, 0xFF34052E, 0xC5855664,
      0x53B02D5D, 0xA99F8FA1, 0x08BA4799, 0x6E85076A };

__constant__ const u32 ks1[256] =
    { 0x4B7A70E9, 0xB5B32944, 0xDB75092E, 0xC4192623, 0xAD6EA6B0, 0x49A7DF7D,
      0x9CEE60B8, 0x8FEDB266, 0xECAA8C71, 0x699A17FF, 0x5664526C, 0xC2B19EE1,
      0x193602A5, 0x75094C29, 0xA0591340, 0xE4183A3E, 0x3F54989A, 0x5B429D65,
      0x6B8FE4D6, 0x99F73FD6, 0xA1D29C07, 0xEFE830F5, 0x4D2D38E6, 0xF0255DC1,
      0x4CDD2086, 0x8470EB26, 0x6382E9C6, 0x021ECC5E, 0x09686B3F, 0x3EBAEFC9,
      0x3C971814, 0x6B6A70A1, 0x687F3584, 0x52A0E286, 0xB79C5305, 0xAA500737,
      0x3E07841C, 0x7FDEAE5C, 0x8E7D44EC, 0x5716F2B8, 0xB03ADA37, 0xF0500C0D,
      0xF01C1F04, 0x0200B3FF, 0xAE0CF51A, 0x3CB574B2, 0x25837A58, 0xDC0921BD,
      0xD19113F9, 0x7CA92FF6, 0x94324773, 0x22F54701, 0x3AE5E581, 0x37C2DADC,
      0xC8B57634, 0x9AF3DDA7, 0xA9446146, 0x0FD0030E, 0xECC8C73E, 0xA4751E41,
      0xE238CD99, 0x3BEA0E2F, 0x3280BBA1, 0x183EB331, 0x4E548B38, 0x4F6DB908,
      0x6F420D03, 0xF60A04BF, 0x2CB81290, 0x24977C79, 0x5679B072, 0xBCAF89AF,
      0xDE9A771F, 0xD9930810, 0xB38BAE12, 0xDCCF3F2E, 0x5512721F, 0x2E6B7124,
      0x501ADDE6, 0x9F84CD87, 0x7A584718, 0x7408DA17, 0xBC9F9ABC, 0xE94B7D8C,
      0xEC7AEC3A, 0xDB851DFA, 0x63094366, 0xC464C3D2, 0xEF1C1847, 0x3215D908,
      0xDD433B37, 0x24C2BA16, 0x12A14D43, 0x2A65C451, 0x50940002, 0x133AE4DD,
      0x71DFF89E, 0x10314E55, 0x81AC77D6, 0x5F11199B, 0x043556F1, 0xD7A3C76B,
      0x3C11183B, 0x5924A509, 0xF28FE6ED, 0x97F1FBFA, 0x9EBABF2C, 0x1E153C6E,
      0x86E34570, 0xEAE96FB1, 0x860E5E0A, 0x5A3E2AB3, 0x771FE71C, 0x4E3D06FA,
      0x2965DCB9, 0x99E71D0F, 0x803E89D6, 0x5266C825, 0x2E4CC978, 0x9C10B36A,
      0xC6150EBA, 0x94E2EA78, 0xA5FC3C53, 0x1E0A2DF4, 0xF2F74EA7, 0x361D2B3D,
      0x1939260F, 0x19C27960, 0x5223A708, 0xF71312B6, 0xEBADFE6E, 0xEAC31F66,
      0xE3BC4595, 0xA67BC883, 0xB17F37D1, 0x018CFF28, 0xC332DDEF, 0xBE6C5AA5,
      0x65582185, 0x68AB9802, 0xEECEA50F, 0xDB2F953B, 0x2AEF7DAD, 0x5B6E2F84,
      0x1521B628, 0x29076170, 0xECDD4775, 0x619F1510, 0x13CCA830, 0xEB61BD96,
      0x0334FE1E, 0xAA0363CF, 0xB5735C90, 0x4C70A239, 0xD59E9E0B, 0xCBAADE14,
      0xEECC86BC, 0x60622CA7, 0x9CAB5CAB, 0xB2F3846E, 0x648B1EAF, 0x19BDF0CA,
      0xA02369B9, 0x655ABB50, 0x40685A32, 0x3C2AB4B3, 0x319EE9D5, 0xC021B8F7,
      0x9B540B19, 0x875FA099, 0x95F7997E, 0x623D7DA8, 0xF837889A, 0x97E32D77,
      0x11ED935F, 0x16681281, 0x0E358829, 0xC7E61FD6, 0x96DEDFA1, 0x7858BA99,
      0x57F584A5, 0x1B227263, 0x9B83C3FF, 0x1AC24696, 0xCDB30AEB, 0x532E3054,
      0x8FD948E4, 0x6DBC3128, 0x58EBF2EF, 0x34C6FFEA, 0xFE28ED61, 0xEE7C3C73,
      0x5D4A14D9, 0xE864B7E3, 0x42105D14, 0x203E13E0, 0x45EEE2B6, 0xA3AAABEA,
      0xDB6C4F15, 0xFACB4FD0, 0xC742F442, 0xEF6ABBB5, 0x654F3B1D, 0x41CD2105,
      0xD81E799E, 0x86854DC7, 0xE44B476A, 0x3D816250, 0xCF62A1F2, 0x5B8D2646,
      0xFC8883A0, 0xC1C7B6A3, 0x7F1524C3, 0x69CB7492, 0x47848A0B, 0x5692B285,
      0x095BBF00, 0xAD19489D, 0x1462B174, 0x23820E00, 0x58428D2A, 0x0C55F5EA,
      0x1DADF43E, 0x233F7061, 0x3372F092, 0x8D937E41, 0xD65FECF1, 0x6C223BDB,
      0x7CDE3759, 0xCBEE7460, 0x4085F2A7, 0xCE77326E, 0xA6078084, 0x19F8509E,
      0xE8EFD855, 0x61D99735, 0xA969A7AA, 0xC50C06C2, 0x5A04ABFC, 0x800BCADC,
      0x9E447A2E, 0xC3453484, 0xFDD56705, 0x0E1E9EC9, 0xDB73DBD3, 0x105588CD,
      0x675FDA79, 0xE3674340, 0xC5C43465, 0x713E38D8, 0x3D28F89E, 0xF16DFF20,
      0x153E21E7, 0x8FB03D4A, 0xE6E39F2B, 0xDB83ADF7 };

__constant__ const u32 ks2[256] =
    { 0xE93D5A68, 0x948140F7, 0xF64C261C, 0x94692934, 0x411520F7, 0x7602D4F7,
      0xBCF46B2E, 0xD4A20068, 0xD4082471, 0x3320F46A, 0x43B7D4B7, 0x500061AF,
      0x1E39F62E, 0x97244546, 0x14214F74, 0xBF8B8840, 0x4D95FC1D, 0x96B591AF,
      0x70F4DDD3, 0x66A02F45, 0xBFBC09EC, 0x03BD9785, 0x7FAC6DD0, 0x31CB8504,
      0x96EB27B3, 0x55FD3941, 0xDA2547E6, 0xABCA0A9A, 0x28507825, 0x530429F4,
      0x0A2C86DA, 0xE9B66DFB, 0x68DC1462, 0xD7486900, 0x680EC0A4, 0x27A18DEE,
      0x4F3FFEA2, 0xE887AD8C, 0xB58CE006, 0x7AF4D6B6, 0xAACE1E7C, 0xD3375FEC,
      0xCE78A399, 0x406B2A42, 0x20FE9E35, 0xD9F385B9, 0xEE39D7AB, 0x3B124E8B,
      0x1DC9FAF7, 0x4B6D1856, 0x26A36631, 0xEAE397B2, 0x3A6EFA74, 0xDD5B4332,
      0x6841E7F7, 0xCA7820FB, 0xFB0AF54E, 0xD8FEB397, 0x454056AC, 0xBA489527,
      0x55533A3A, 0x20838D87, 0xFE6BA9B7, 0xD096954B, 0x55A867BC, 0xA1159A58,
      0xCCA92963, 0x99E1DB33, 0xA62A4A56, 0x3F3125F9, 0x5EF47E1C, 0x9029317C,
      0xFDF8E802, 0x04272F70, 0x80BB155C, 0x05282CE3, 0x95C11548, 0xE4C66D22,
      0x48C1133F, 0xC70F86DC, 0x07F9C9EE, 0x41041F0F, 0x404779A4, 0x5D886E17,
      0x325F51EB, 0xD59BC0D1, 0xF2BCC18F, 0x41113564, 0x257B7834, 0x602A9C60,
      0xDFF8E8A3, 0x1F636C1B, 0x0E12B4C2, 0x02E1329E, 0xAF664FD1, 0xCAD18115,
      0x6B2395E0, 0x333E92E1, 0x3B240B62, 0xEEBEB922, 0x85B2A20E, 0xE6BA0D99,
      0xDE720C8C, 0x2DA2F728, 0xD0127845, 0x95B794FD, 0x647D0862, 0xE7CCF5F0,
      0x5449A36F, 0x877D48FA, 0xC39DFD27, 0xF33E8D1E, 0x0A476341, 0x992EFF74,
      0x3A6F6EAB, 0xF4F8FD37, 0xA812DC60, 0xA1EBDDF8, 0x991BE14C, 0xDB6E6B0D,
      0xC67B5510, 0x6D672C37, 0x2765D43B, 0xDCD0E804, 0xF1290DC7, 0xCC00FFA3,
      0xB5390F92, 0x690FED0B, 0x667B9FFB, 0xCEDB7D9C, 0xA091CF0B, 0xD9155EA3,
      0xBB132F88, 0x515BAD24, 0x7B9479BF, 0x763BD6EB, 0x37392EB3, 0xCC115979,
      0x8026E297, 0xF42E312D, 0x6842ADA7, 0xC66A2B3B, 0x12754CCC, 0x782EF11C,
      0x6A124237, 0xB79251E7, 0x06A1BBE6, 0x4BFB6350, 0x1A6B1018, 0x11CAEDFA,
      0x3D25BDD8, 0xE2E1C3C9, 0x44421659, 0x0A121386, 0xD90CEC6E, 0xD5ABEA2A,
      0x64AF674E, 0xDA86A85F, 0xBEBFE988, 0x64E4C3FE, 0x9DBC8057, 0xF0F7C086,
      0x60787BF8, 0x6003604D, 0xD1FD8346, 0xF6381FB0, 0x7745AE04, 0xD736FCCC,
      0x83426B33, 0xF01EAB71, 0xB0804187, 0x3C005E5F, 0x77A057BE, 0xBDE8AE24,
      0x55464299, 0xBF582E61, 0x4E58F48F, 0xF2DDFDA2, 0xF474EF38, 0x8789BDC2,
      0x5366F9C3, 0xC8B38E74, 0xB475F255, 0x46FCD9B9, 0x7AEB2661, 0x8B1DDF84,
      0x846A0E79, 0x915F95E2, 0x466E598E, 0x20B45770, 0x8CD55591, 0xC902DE4C,
      0xB90BACE1, 0xBB8205D0, 0x11A86248, 0x7574A99E, 0xB77F19B6, 0xE0A9DC09,
      0x662D09A1, 0xC4324633, 0xE85A1F02, 0x09F0BE8C, 0x4A99A025, 0x1D6EFE10,
      0x1AB93D1D, 0x0BA5A4DF, 0xA186F20F, 0x2868F169, 0xDCB7DA83, 0x573906FE,
      0xA1E2CE9B, 0x4FCD7F52, 0x50115E01, 0xA70683FA, 0xA002B5C4, 0x0DE6D027,
      0x9AF88C27, 0x773F8641, 0xC3604C06, 0x61A806B5, 0xF0177A28, 0xC0F586E0,
      0x006058AA, 0x30DC7D62, 0x11E69ED7, 0x2338EA63, 0x53C2DD94, 0xC2C21634,
      0xBBCBEE56, 0x90BCB6DE, 0xEBFC7DA1, 0xCE591D76, 0x6F05E409, 0x4B7C0188,
      0x39720A3D, 0x7C927C24, 0x86E3725F, 0x724D9DB9, 0x1AC15BB4, 0xD39EB8FC,
      0xED545578, 0x08FCA5B5, 0xD83D7CD3, 0x4DAD0FC4, 0x1E50EF5E, 0xB161E6F8,
      0xA28514D9, 0x6C51133C, 0x6FD5C7E7, 0x56E14EC4, 0x362ABFCE, 0xDDC6C837,
      0xD79A3234, 0x92638212, 0x670EFA8E, 0x406000E0 };

__constant__ const u32 ks3[256] =
    { 0x3A39CE37, 0xD3FAF5CF, 0xABC27737, 0x5AC52D1B, 0x5CB0679E, 0x4FA33742,
      0xD3822740, 0x99BC9BBE, 0xD5118E9D, 0xBF0F7315, 0xD62D1C7E, 0xC700C47B,
      0xB78C1B6B, 0x21A19045, 0xB26EB1BE, 0x6A366EB4, 0x5748AB2F, 0xBC946E79,
      0xC6A376D2, 0x6549C2C8, 0x530FF8EE, 0x468DDE7D, 0xD5730A1D, 0x4CD04DC6,
      0x2939BBDB, 0xA9BA4650, 0xAC9526E8, 0xBE5EE304, 0xA1FAD5F0, 0x6A2D519A,
      0x63EF8CE2, 0x9A86EE22, 0xC089C2B8, 0x43242EF6, 0xA51E03AA, 0x9CF2D0A4,
      0x83C061BA, 0x9BE96A4D, 0x8FE51550, 0xBA645BD6, 0x2826A2F9, 0xA73A3AE1,
      0x4BA99586, 0xEF5562E9, 0xC72FEFD3, 0xF752F7DA, 0x3F046F69, 0x77FA0A59,
      0x80E4A915, 0x87B08601, 0x9B09E6AD, 0x3B3EE593, 0xE990FD5A, 0x9E34D797,
      0x2CF0B7D9, 0x022B8B51, 0x96D5AC3A, 0x017DA67D, 0xD1CF3ED6, 0x7C7D2D28,
      0x1F9F25CF, 0xADF2B89B, 0x5AD6B472, 0x5A88F54C, 0xE029AC71, 0xE019A5E6,
      0x47B0ACFD, 0xED93FA9B, 0xE8D3C48D, 0x283B57CC, 0xF8D56629, 0x79132E28,
      0x785F0191, 0xED756055, 0xF7960E44, 0xE3D35E8C, 0x15056DD4, 0x88F46DBA,
      0x03A16125, 0x0564F0BD, 0xC3EB9E15, 0x3C9057A2, 0x97271AEC, 0xA93A072A,
      0x1B3F6D9B, 0x1E6321F5, 0xF59C66FB, 0x26DCF319, 0x7533D928, 0xB155FDF5,
      0x03563482, 0x8ABA3CBB, 0x28517711, 0xC20AD9F8, 0xABCC5167, 0xCCAD925F,
      0x4DE81751, 0x3830DC8E, 0x379D5862, 0x9320F991, 0xEA7A90C2, 0xFB3E7BCE,
      0x5121CE64, 0x774FBE32, 0xA8B6E37E, 0xC3293D46, 0x48DE5369, 0x6413E680,
      0xA2AE0810, 0xDD6DB224, 0x69852DFD, 0x09072166, 0xB39A460A, 0x6445C0DD,
      0x586CDECF, 0x1C20C8AE, 0x5BBEF7DD, 0x1B588D40, 0xCCD2017F, 0x6BB4E3BB,
      0xDDA26A7E, 0x3A59FF45, 0x3E350A44, 0xBCB4CDD5, 0x72EACEA8, 0xFA6484BB,
      0x8D6612AE, 0xBF3C6F47, 0xD29BE463, 0x542F5D9E, 0xAEC2771B, 0xF64E6370,
      0x740E0D8D, 0xE75B1357, 0xF8721671, 0xAF537D5D, 0x4040CB08, 0x4EB4E2CC,
      0x34D2466A, 0x0115AF84, 0xE1B00428, 0x95983A1D, 0x06B89FB4, 0xCE6EA048,
      0x6F3F3B82, 0x3520AB82, 0x011A1D4B, 0x277227F8, 0x611560B1, 0xE7933FDC,
      0xBB3A792B, 0x344525BD, 0xA08839E1, 0x51CE794B, 0x2F32C9B7, 0xA01FBAC9,
      0xE01CC87E, 0xBCC7D1F6, 0xCF0111C3, 0xA1E8AAC7, 0x1A908749, 0xD44FBD9A,
      0xD0DADECB, 0xD50ADA38, 0x0339C32A, 0xC6913667, 0x8DF9317C, 0xE0B12B4F,
      0xF79E59B7, 0x43F5BB3A, 0xF2D519FF, 0x27D9459C, 0xBF97222C, 0x15E6FC2A,
      0x0F91FC71, 0x9B941525, 0xFAE59361, 0xCEB69CEB, 0xC2A86459, 0x12BAA8D1,
      0xB6C1075E, 0xE3056A0C, 0x10D25065, 0xCB03A442, 0xE0EC6E0E, 0x1698DB3B,
      0x4C98A0BE, 0x3278E964, 0x9F1F9532, 0xE0D392DF, 0xD3A0342B, 0x8971F21E,
      0x1B0A7441, 0x4BA3348C, 0xC5BE7120, 0xC37632D8, 0xDF359F8D, 0x9B992F2E,
      0xE60B6F47, 0x0FE3F11D, 0xE54CDA54, 0x1EDAD891, 0xCE6279CF, 0xCD3E7E6F,
      0x1618B166, 0xFD2C1D05, 0x848FD2C5, 0xF6FB2299, 0xF523F357, 0xA6327623,
      0x93A83531, 0x56CCCD02, 0xACF08162, 0x5A75EBB5, 0x6E163697, 0x88D273CC,
      0xDE966292, 0x81B949D0, 0x4C50901B, 0x71C65614, 0xE6C6C7BD, 0x327A140A,
      0x45E1D006, 0xC3F27B9A, 0xC9AA53FD, 0x62A80F00, 0xBB25BFE2, 0x35BDD2F6,
      0x71126905, 0xB2040222, 0xB6CBCF7C, 0xCD769C2B, 0x53113EC0, 0x1640E3D3,
      0x38ABBD60, 0x2547ADF0, 0xBA38209C, 0xF746CE76, 0x77AFA1C5, 0x20756060,
      0x85CBFE4E, 0x8AE88DD8, 0x7AAAF9B0, 0x4CF9AA7E, 0x1948C25C, 0x02FB8A8C,
      0x01C36AE4, 0xD6EBE1F9, 0x90D4F869, 0xA65CDEA0, 0x3F09252D, 0xC208E69F,
      0xB74E6132, 0xCE77E25B, 0x578FDFE3, 0x3AC372E6 };

__constant__ const u32 ps[N_BLOWFISH_ROUNDS + 2] =
    { 0x243F6A88, 0x85A308D3, 0x13198A2E, 0x03707344, 0xA4093822, 0x299F31D0,
      0x082EFA98, 0xEC4E6C89, 0x452821E6, 0x38D01377, 0xBE5466CF, 0x34E90C6C,
      0xC0AC29B7, 0xC97C50DD, 0x3F84D5B5, 0xB5470917, 0x9216D5D9, 0x8979FB1B };

#define F(x)                                            \
  (((subkey.s0[((u8*)&x)[3]] + subkey.s1[((u8*)&x)[2]]) \
    ^ subkey.s2[((u8*)&x)[1]])                          \
   + subkey.s3[((u8*)&x)[0]])
#define R(l, r, i)    \
  do {                \
    l ^= subkey.p[i]; \
    r ^= F(l);        \
  } while (0)

__device__ void BlowfishCreateSubkey(
    BlowfishSubkey& subkey_out, u8* key, u32 key_len)
{
  u32 i;
  u32 j;
  u32 data;
  u32 left;
  u32 right;

  for (i = 0; i < N_BLOWFISH_ROUNDS + 2; i++) {
    subkey_out.p[i] = ps[i];
  }
  for (i = 0; i < 256; i++) {
    subkey_out.s0[i] = ks0[i];
    subkey_out.s1[i] = ks1[i];
    subkey_out.s2[i] = ks2[i];
    subkey_out.s3[i] = ks3[i];
  }

  for (i = j = 0; i < N_BLOWFISH_ROUNDS + 2; i++) {
    data = key[j] << 24 | (key[(j + 1) % key_len]) << 16
           | (key[(j + 2) % key_len]) << 8 | (key[(j + 3) % key_len]);
    subkey_out.p[i] ^= data;
    j = (j + 4) % key_len;
  }

  left = right = 0;
  for (i = 0; i < N_BLOWFISH_ROUNDS + 2; i += 2) {
    BlowfishEncrypt(subkey_out, &left, &right);
    subkey_out.p[i] = left;
    subkey_out.p[i + 1] = right;
  }
  // TODO: Put all in one loop?
  for (i = 0; i < 256; i += 2) {
    BlowfishEncrypt(subkey_out, &left, &right);
    subkey_out.s0[i] = left;
    subkey_out.s0[i + 1] = right;
  }
  for (i = 0; i < 256; i += 2) {
    BlowfishEncrypt(subkey_out, &left, &right);
    subkey_out.s1[i] = left;
    subkey_out.s1[i + 1] = right;
  }
  for (i = 0; i < 256; i += 2) {
    BlowfishEncrypt(subkey_out, &left, &right);
    subkey_out.s2[i] = left;
    subkey_out.s2[i + 1] = right;
  }
  for (i = 0; i < 256; i += 2) {
    BlowfishEncrypt(subkey_out, &left, &right);
    subkey_out.s3[i] = left;
    subkey_out.s3[i + 1] = right;
  }
}

__device__ void BlowfishEncrypt(
    BlowfishSubkey& subkey, u32* left_out, u32* right_out)
{
  u32 left(*left_out);
  u32 right(*right_out);

  R(left, right, 0);
  R(right, left, 1);
  R(left, right, 2);
  R(right, left, 3);
  R(left, right, 4);
  R(right, left, 5);
  R(left, right, 6);
  R(right, left, 7);
  R(left, right, 8);
  R(right, left, 9);
  R(left, right, 10);
  R(right, left, 11);
  R(left, right, 12);
  R(right, left, 13);
  R(left, right, 14);
  R(right, left, 15);

  left ^= subkey.p[N_BLOWFISH_ROUNDS];
  right ^= subkey.p[N_BLOWFISH_ROUNDS + 1];

  *left_out = right;
  *right_out = left;
}

// Not in use.
//__device__ void BlowfishDecrypt(BlowfishSubkey& subkey, u32* left_out, u32*
//right_out)
//{
//  u32 left(*left_out);
//  u32 right(*right_out);
//
//  R(left, right, 17); R(right, left, 16); R(left, right, 15); R(right, left,
//  14);
//  R(left, right, 13); R(right, left, 12); R(left, right, 11); R(right, left,
//  10);
//  R(left, right, 9); R(right, left, 8); R(left, right, 7); R(right, left, 6);
//  R(left, right, 5); R(right, left, 4); R(left, right, 3); R(right, left, 2);
//
//  left ^= subkey.p[1];
//  right ^= subkey.p[0];
//
//  *left_out = right;
//  *right_out = left;
//}

__device__ void BlowfishDecryptBufferCFB(
    BlowfishSubkey& subkey, u32* buf_out, u32* buf_in, u32 n_blocks, u32* iv)
{
  u32 d1(iv[0]);
  u32 d2(iv[1]);

  for (int i(0); i < n_blocks * 2; i += 2) {
    BlowfishEncrypt(subkey, &d1, &d2);

    u32 x1(SwapEndian32(buf_in[i]));
    u32 x2(SwapEndian32(buf_in[i + 1]));

    buf_out[i] = SwapEndian32(x1 ^ d1);
    buf_out[i + 1] = SwapEndian32(x2 ^ d2);

    d1 = x1;
    d2 = x2;
  }
}

///////////////////////////////////////////////////////////////////////////////
// Util, device

// Inner loop: YES
__device__ u32 SwapEndian32(u32 x)
{
  return __byte_perm(x, 0, 0x0123);
}

__device__ void SwapEndian32Ptr(u32* x)
{
  *x = __byte_perm(*x, 0, 0x0123);
}

// Inner loop: NO
__device__ u32 Min(u32 a, u32 b)
{
  return a < b ? a : b;
}

///////////////////////////////////////////////////////////////////////////////
// Util, host

// Inner loop: NO
__host__ void HostSwapEndian32Ptr(u32* x)
{
  u32 a(*x);
  *x =
      (a << 24) | (a >> 24) | ((a & 0x0000ff00) << 8) | ((a & 0x00ff0000) >> 8);
}

// Inner loop: NO
u32v u8vToLittleEndianu32v(u8v& in)
{
  u32v out;
  for (u8v::iterator iter(in.begin()); iter != in.end(); iter += 4) {
    u32* in32(reinterpret_cast<u32*>(&*iter));
    HostSwapEndian32Ptr(in32);
    out.push_back(*in32);
  }
  return out;
}
